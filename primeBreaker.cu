#include "hip/hip_runtime.h"
#include "primeBreaker.hpp"



__device__ void lock(int *mutex) {while (atomicCAS(mutex, 0, 1) != 0);}
__device__ void unlock(int *mutex) {	atomicExch(mutex, 0); }



/** \brief
    je suis la fonction qui permet de decomposeur un numero en facteurs premiers
*/
__global__ void facGPU(uint64_t  N,uint64_t *const dev_primes,cell *const dev_facteurs,int *mutex)
{
      int gid = threadIdx.x +  blockIdx.x*blockDim.x;

      while(gid<N)
      {

              while (N!=1)
              {
                      if(dev_primes[gid]!=0)
                      {
                          printf(" N= [%lld] val[%lld] \n",N,dev_primes[gid]);
                          if(N%dev_primes[gid]==0)
                          {
                                dev_facteurs[gid].expo+=1;
                              //  bool leave = true;
                                //while (leave)
                                {
                                  if (atomicCAS(mutex, 0, 1) == 0)
                                  {
                                    N=N/dev_primes[gid];
                                    //leave = false;
                                    atomicExch(mutex, 0);
                                  }
                                  //break;
                                }
                                //lock(mutex);

                                //unlock(mutex);
                           }
                      }
                }

            gid+=blockDim.x * gridDim.x;

          }





}
