#include "hip/hip_runtime.h"
#include "primeBreaker.hpp"


/**	\brief Je suis une fonction d'évaluation partielle de la primalité d'un nombre premier.
  */
__global__
void isPrime(
		uint64_t *possibles_premiers,
		unsigned int *res_operations,
		uint64_t N,
		uint64_t sqrtN
		){

	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	int initial_gid = gid;
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	extern __shared__ unsigned int cache[];

	cache[tid] = 1;
	while (gid < sqrtN){
		cache[tid] = (N%possibles_premiers[gid] != 0); // Si il n'y a pas de reste (le nombre est divisé entièrement par un autre nombre) j'inscrit zero dans le cache 

		__syncthreads();

		int offset = blockDim.x/2;
		while (offset > 1) {
			if (tid < offset) {
				cache[tid] = umin ( cache[tid], cache[tid+offset] );
			}
			__syncthreads();
			offset /=2;
		}
		
		if (tid == 0) { res_operations[bid] = cache[0]; }
		
		gid += gridDim.x * blockDim.x;	
	}

	if (initial_gid < gridDim.x)
		res_operations[0] = ((res_operations[0] != 0) && (res_operations[initial_gid] != 0));
}

/*
__global__ void searchPrimeGPU(
		uint64_t *const dev_possiblesPremiers,
		uint64_t  *const dev_primes,
		uint64_t const limit,
		int const taille
		)
{
    int gid = threadIdx.x + blockIdx.x*blockDim.x;

    while(gid < taille)
    {

        //isPrimeGPU(dev_tab_possibles_diviseurs,dev_resOperations,N,taille);

      //  dev_primes[gid]=dev_resOperations[0];
        gid+=blockDim.x*gridDim.x;
    }

}
*/

/** \brief
    je suis la fonction qui permet de decomposeur un numero en facteurs premiers
*/
__global__ 
void facGPU(
		uint64_t  N,
		uint64_t *const dev_primes,
		cell *const dev_facteurs
)
{
}
