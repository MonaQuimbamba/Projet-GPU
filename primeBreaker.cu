#include "hip/hip_runtime.h"
#include "primeBreaker.hpp"


/** \brief
    je suis la fonction qui permet de decomposeur un numero en facteurs premiers
*/
__global__ void facGPU(uint64_t const N,uint64_t *const dev_primes,cell *const dev_facteurs)
{
      int gid = threadIdx.x +  blockIdx.x*blockDim.x;

      while( gid < N)
      {
            if(N%dev_primes[gid]==0)
            {
                  facteurs[gid]->expo+=1;
            }

            gid+=blockDim.x * gridDim.x;
      }


}
