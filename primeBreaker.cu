#include "hip/hip_runtime.h"
#include "primeBreaker.hpp"


__global__ void isPrimeGPU(uint64_t *const dev_N,unsigned int  *const isPrime, uint64_t const N)
{
       int global_t_id = threadIdx.x +  blockIdx.x*blockDim.x;
      int t_id = threadIdx.x;
      extern __shared__ unsigned int cache[];
      while( global_t_id < N)
      {
          	cache[t_id]=  (N%dev_N[global_t_id])==0 ? 0 : 1;
            //printf("%d le val , le cache val %d \n",dev_N[global_t_id],cache[t_id] );
            global_t_id+=blockDim.x * gridDim.x;
        }

      	__syncthreads();
      /*  if(threadIdx.x==0)
        {
          for(int i=0 ; i< blockDim.x;i++)
          {
            printf(" le cache %d , le idThreads %d \n",cache[i],i );
          }
        }*/
       unsigned int i = 1;
      	while ( i < blockDim.x )
      	{

            int id = 2 * i * t_id;
        		if ( id < blockDim.x )
        		{

                  printf(" le cache %d , le idThreads %d \n",cache[id],i );
                //if(id+i<N)
                {
                    //printf("%d ",cache[id] );
        			     cache[id] =umin( cache[id], cache[id + i] );
                }
        		}
        		__syncthreads();
        		i *= 2;



      	}
        /*unsigned int i = blockDim.x/2;
      	while(i!=0)
      	{
      		 if(t_id < i)
      		 {
      			 cache[t_id]=umin( cache[t_id], cache[t_id + i] );
      		 }
      		 __syncthreads();
      		 i/=2;
      	}*/
        if(threadIdx.x==0) {
          isPrime[blockIdx.x] = cache[0];
          //printf("%d ",isPrime[blockIdx.x] );
        }

}
