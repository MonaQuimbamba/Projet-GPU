#include "hip/hip_runtime.h"
#include "primeBreaker.hpp"

__device__ uint64_t atomicExch_d(uint64_t* address, uint64_t val)
{
  uint64_t old = *address;

 do{
    *address = val;

  }while (old==val);

  return old;
}

/** \brief
      je suis la focntion qui verifie la primalité d'un numero ,
      exemple d'éxecution : chaque thread verifie un numero inférieu à N peut diviser N
      si oui on ajout 0 dans le tableu de la memoire partage sinon on ajoute 1 , et au final on fait
      une reduction pour verifier si y'a des 0 dans la memoire partagée et on retourne un tableu de la taille
      d'un block avec que des 0 et 1 et dans CPU on verifie si ya des 0 c-à-d qu'il n'est pas premier sinon oui
*/
__global__ void isPrimeGPU(uint64_t *const dev_tab_possibles_diviseurs,unsigned int  *const dev_resOperations, uint64_t const N,size_t const taille)
{
      int gid = threadIdx.x +  blockIdx.x*blockDim.x;
      int tid = threadIdx.x;
      extern __shared__ unsigned int cache[];
      cache[tid]= 1;
      while( gid < taille)
      {

          if((N%dev_tab_possibles_diviseurs[gid])==0 )
            cache[tid]= 0;
          gid+=blockDim.x * gridDim.x;
      }

      // une reduction sur le cache pour trouver  le mimimun du cache
      __syncthreads();
      unsigned int i = blockDim.x/2;
      	while(i!=0)
      	{
      		 if(tid < i)
      		 {
      			 cache[tid]=umin( cache[tid], cache[tid + i] );
      		 }
      		 __syncthreads();
      		 i/=2;
      	}
       if(threadIdx.x==0)  dev_resOperations[blockIdx.x] = cache[0];

     // une reduction final, sur le tableau des operations
       __syncthreads();
       unsigned int j = blockDim.x/2;
        while(i!=0)
        {
           if(tid < j)
           {   dev_resOperations[tid]=umin( dev_resOperations[tid], dev_resOperations[tid + j] );}
           __syncthreads();
           j/=2;
        }
}

__global__ void searchPrimeGPU(uint64_t *const dev_possiblesPremiers,uint64_t  *const dev_primes,uint64_t const limiter, int const taille)
{
    int gid = threadIdx.x + blockIdx.x*blockDim.x;

    while(gid < taille)
    {

        isPrimeGPU(dev_tab_possibles_diviseurs,dev_resOperations,N,taille);

        dev_primes[gid]=dev_resOperations[0];
        gid+=blockDim.x*gridDim.x;
    }

}


/** \brief
    je suis la fonction qui permet de decomposeur un numero en facteurs premiers
*/
__global__ void facGPU(uint64_t  N,uint64_t *const dev_primes,cell *const dev_facteurs,int *mutex)
{
    int gid = threadIdx.x +  blockIdx.x*blockDim.x;

    while(gid<N)
    {

        while (N!=1)
        {
            if(dev_primes[gid]!=0)
            {
                printf(" N= [%lld] val[%lld] \n",N,dev_primes[gid]);
                if(N%dev_primes[gid]==0)
                {
                    dev_facteurs[gid].expo+=1;
                    //  bool leave = true;
                    //while (leave)
                    {
                        if (atomicCAS(mutex, 0, 1) == 0)
                        {
                            N=N/dev_primes[gid];
                            //leave = false;
                            atomicExch(mutex, 0);
                        }
                        //break;
                    }
                    //lock(mutex);

                    //unlock(mutex);
                }
            }
        }

        gid+=blockDim.x * gridDim.x;

    }





}