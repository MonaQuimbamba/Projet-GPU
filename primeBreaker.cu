#include "hip/hip_runtime.h"

#include "primeBreaker.hpp"
#include <bits/stdc++.h>

__global__ void isPrimeGPU(uint64_t *const dev_N,const uint64_t N,unsigned int  *const isPrime)
{
        int global_id = threadIdx.x +  blockIdx.x*blockDim.x;
      	int t_id = threadIdx.x;
      	extern __shared__ unsigned int cache[];
      	cache[t_id]= global_id < N ? (floor(N/dev_N[global_id]) == N/dev_N[global_id]) : 0; // ajouter 2 au cas où
      	__syncthreads();
      	unsigned int i = blockDim.x/2;
      	while(i!=0)
      	{
      		 if(t_id < i)
      		 {
      			  cache[t_id]=umax( cache[t_id], cache[t_id + i] );
      		 }
      		 __syncthreads();
      		 i/=2;
      	}

      	if(t_id==0) isPrime[blockIdx.x] = cache[0];
}
