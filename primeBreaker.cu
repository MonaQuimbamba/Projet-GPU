#include "hip/hip_runtime.h"
#include "primeBreaker.hpp"

//__device__   int global_var;

/** \brief
      je suis la fonction qui verifie la primalité d'un numero ,
      exemple d'éxecution : chaque thread verifie un numero inférieu à N peut diviser N
      si oui on ajout 0 dans le tableu de la memoire partage sinon on ajoute 1 , et au final on fait
      une reduction pour verifier si y'a des 0 dans la memoire partagée et on retourne un tableu de la taille
      d'un block avec que des 0 et 1 et dans CPU on verifie si ya des 0 c-à-d qu'il n'est pas premier sinon oui
*/
__global__ void isPrimeGPU(uint64_t *const dev_N, unsigned int  *const isPrime, uint64_t const N, size_t const taille)
{
      int global_t_id = threadIdx.x +  blockIdx.x*blockDim.x;
      int t_id = threadIdx.x;
      extern __shared__ unsigned int cache[];
      cache[t_id]= 1;
      while( global_t_id < taille)
      {
          if((N%dev_N[global_t_id])==0 )
            cache[t_id]= 0;
          global_t_id+=blockDim.x * gridDim.x;
      }
      __syncthreads();
      unsigned int i = blockDim.x/2;
      while(i!=0)
      {
      		 if(t_id < i)
      		 {
      			 cache[t_id]=umin( cache[t_id], cache[t_id + i] );
      		 }
      		 __syncthreads();
      		 i/=2;
      }
      if(threadIdx.x==0)  isPrime[blockIdx.x] = cache[0];


}


__global__ void searchPrimeGPU(uint64_t *const dev_N,uint64_t  *const primesNumbers,uint64_t const N, int const taille)
{
      int global_t_id = threadIdx.x + blockIdx.x*blockDim.x;

      while(global_t_id < taille)
      {

            //launchKernelIsPrimeGPU<0>(dev_N[global_t_id],isPrime);
            if(N%dev_N[global_t_id]==0)
            {
                /*  unsigned int isPrime=1;
                    for (uint64_t val = dev_N[global_t_id] ;val >= 2; val-=1)
                    {
                        if (dev_N[global_t_id]%val== 0)
                        {isPrime=0;}
                    }
                    if(isPrime==1)*/
                      primesNumbers[global_t_id]=dev_N[global_t_id];
            }


            global_t_id+=blockDim.x*gridDim.x;
      }

}
