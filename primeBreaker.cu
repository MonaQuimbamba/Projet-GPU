#include "hip/hip_runtime.h"

#include "primeBreaker.hpp"
#include <bits/stdc++.h>

__global__ void isPrimeGPU(uint64_t *const dev_N,unsigned int  *const isPrime, uint64_t const N)
{
       int global_id = threadIdx.x +  blockIdx.x*blockDim.x;
      int t_id = threadIdx.x;
      extern __shared__ unsigned int cache[];
      while( global_id < N)
      {

          	cache[t_id]= N%dev_N[global_id] == 0 ? 0 : 1;

            global_id+=blockDim.x * gridDim.x;
        }

      	__syncthreads();
       unsigned int i = 1;
      	while ( i < blockDim.x )
      	{

      		int id = 2 * i * t_id;
      		if ( id < blockDim.x )
      		{

      			cache[id] =umin( cache[id], cache[id + i] );
      		}
      		__syncthreads();
      		i *= 2;
      	}
        /*unsigned int i = blockDim.x/2;
      	while(i!=0)
      	{
      		 if(t_id < i)
      		 {
      			 cache[t_id]=umin( cache[t_id], cache[t_id + i] );
      		 }
      		 __syncthreads();
      		 i/=2;
      	}*/
        if(threadIdx.x==0) isPrime[blockIdx.x] = cache[0];

}
