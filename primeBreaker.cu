#include "hip/hip_runtime.h"
#include "primeBreaker.hpp"



__device__ uint64_t atomicExch_d(uint64_t* address, uint64_t val)
{
  uint64_t old = *address;

 do{
    *address = val;

  }while (old==val);

  return old;
}

/** \brief
    je suis la fonction qui permet de decomposeur un numero en facteurs premiers
*/
__global__ void facGPU(uint64_t  N,uint64_t *const dev_primes,cell *const dev_facteurs)
{
      int gid = threadIdx.x +  blockIdx.x*blockDim.x;

      while( gid < N)
      {
            if(N%dev_primes[gid]==0)
            {
              //__threadfence_system();
                  dev_facteurs[gid].expo+=1;
                  //N=N/dev_primes[gid];
                  atomicExch_d(&N,N/dev_primes[gid]);
            }

            gid+=blockDim.x * gridDim.x;
      }


}
