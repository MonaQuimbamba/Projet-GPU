#include "hip/hip_runtime.h"
#include "primeBreaker.hpp"


/**	\brief Je suis une fonction d'évaluation de la primalité d'un nombre premier.
  */
__global__
void isPrime(
		uint64_t *possibles_premiers,
		unsigned int *res_operations,
		uint64_t N,
		uint64_t sqrtN
		){

	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	int initial_gid = gid;
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	extern __shared__ unsigned int cache[];

	cache[tid] = 1;
	while (gid < sqrtN){
		cache[tid] = (N%possibles_premiers[gid] != 0); // Si il n'y a pas de reste (le nombre est divisé entièrement par un autre nombre) j'inscrit zero dans le cache 

		__syncthreads();

		int offset = blockDim.x/2;
		while (offset > 1) {
			if (tid < offset) {
				cache[tid] = umin ( cache[tid], cache[tid+offset] );
			}
			__syncthreads();
			offset /=2;
		}
		
		if (tid == 0) { res_operations[bid] = cache[0]; }
		
		gid += gridDim.x * blockDim.x;	
	}

	if (initial_gid < ((sqrtN+blockDim.x-1)/blockDim.x))
		res_operations[0] = ((res_operations[0] != 0) && (res_operations[initial_gid] != 0));
}

/*	/brief	Je suis une fonction qui récupère les nombres premiers inférieur à une borne renseignée
		à paramètre.
  
 */
__global__ void searchPrimeGPU(
		uint64_t *possibles_premiers,
		uint64_t *square_roots,
		uint64_t borne_sup,
		uint64_t *premiers)
{
	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	/*
	if (gid == 0) {
		printf("Afficher les données initialisées sur le GPU ");
		printf("possibles premiers ");
		for (int i = 0; i < (borne_sup-2); i++){
			printf("[%d]",possibles_premiers[i]);
		}
		printf("\n\n");
		
		printf("square_roots ");
		for (int i = 0; i < (borne_sup-2); i++){
			printf("[%d]",square_roots[i]);
		}
		printf("\n\n");
		
		printf("premiers ");
		for (int i = 0; i < (borne_sup-2); i++){
			printf("[%d]",premiers[i]);
		}
		printf("\n\n");

		printf("Fin affichage des données initialisées\n");	
	}
	*/
	while (gid < borne_sup-2) {
		unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int)*((square_roots[gid]+blockDim.x-1)/blockDim.x));

		isPrime<<<gridDim.x,blockDim.x,blockDim.x*sizeof(unsigned int)>>>
			(possibles_premiers,
			 res_operations,
			 possibles_premiers[gid],
			 square_roots[gid]
			 );
		/*	
		printf("gid = %d\n", gid);
		printf("wid = %d\n", wid);
		printf("square_roots[gid] = %d\n", square_roots[gid]);
		printf("possibles_premiers[gid] = %d\n", possibles_premiers[gid]);
		printf("res_operations[0] = %d\n", res_operations[0]);	
		*/
		
		premiers[gid] = res_operations[0];
		
		free(res_operations);
		gid += gridDim.x * blockDim.x;
		
	}
	
}


/** \brief
    je suis la fonction qui permet de decomposeur un numero en facteurs premiers
*/
__global__ 
void facGPU(
		uint64_t  N,
		uint64_t *const dev_primes,
		cell *const dev_facteurs
)
{
}
