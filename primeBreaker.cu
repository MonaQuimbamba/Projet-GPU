#include "hip/hip_runtime.h"
#include "primeBreaker.hpp"


/** \brief
      je suis la focntion qui verifie la primalité d'un numero ,
      exemple d'éxecution : chaque thread verifie un numero inférieu à N peut diviser N
      si oui on ajout 0 dans le tableu de la memoire partage sinon on ajoute 1 , et au final on fait
      une reduction pour verifier si y'a des 0 dans la memoire partagée et on retourne un tableu de la taille
      d'un block avec que des 0 et 1 et dans CPU on verifie si ya des 0 c-à-d qu'il n'est pas premier sinon oui
*/
__global__ void isPrimeGPU_naif(uint64_t *const dev_N,unsigned int  *const isPrime, uint64_t const N)
{
      int global_t_id = threadIdx.x +  blockIdx.x*blockDim.x;

      while( global_t_id < N)
      {
            isPrime[global_t_id]=((N%dev_N[global_t_id])==0 ) ? 0 : 1;
            global_t_id+=blockDim.x * gridDim.x;
      }


}


/** \brief
    je suis la fonction qui permet de decomposeur un numero en facteurs premiers
*/
__global__ void facGPU(uint64_t *const dev_N,uint64_t  *const facteurs, uint64_t const N, unsigned int const taille)
{
      int global_t_id = threadIdx.x +  blockIdx.x*blockDim.x;

      while( global_t_id < taille)
      {
            facteurs[global_t_id]=((N%dev_N[global_t_id])==0 ) ? dev_N[global_t_id] : 1; // si c'est un facteur premier on l'ajoute dans le tableau sionon on ajoute 1 
            global_t_id+=blockDim.x * gridDim.x;
      }


}
