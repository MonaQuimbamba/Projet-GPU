#include "hip/hip_runtime.h"
#include "primeBreaker.hpp"


/** \brief
      je suis la focntion qui verifie la primalité d'un numero ,
      exemple d'éxecution : chaque thread verifie un numero inférieu à N peut diviser N
      si oui on ajout 0 dans le tableu de la memoire partage sinon on ajoute 1 , et au final on fait
      une reduction pour verifier si y'a des 0 dans la memoire partagée et on retourne un tableu de la taille
      d'un block avec que des 0 et 1 et dans CPU on verifie si ya des 0 c-à-d qu'il n'est pas premier sinon oui
*/
__global__ void isPrimeGPU(uint64_t *const dev_tab_possibles_diviseurs,unsigned int  *const dev_resOperations, uint64_t const N,size_t const taille)
{
      int gid = threadIdx.x +  blockIdx.x*blockDim.x;
      int tid = threadIdx.x;
      extern __shared__ unsigned int cache[];
      cache[tid]= 1;
      while( gid < taille)
      {

          if((N%dev_tab_possibles_diviseurs[gid])==0 )
            cache[tid]= 0;
          gid+=blockDim.x * gridDim.x;
      }

      // une reduction sur le cache pour trouver  le mimimun du cache 
      __syncthreads();
      unsigned int i = blockDim.x/2;
      	while(i!=0)
      	{
      		 if(tid < i)
      		 {
      			 cache[tid]=umin( cache[tid], cache[tid + i] );
      		 }
      		 __syncthreads();
      		 i/=2;
      	}
       if(threadIdx.x==0)  dev_resOperations[blockIdx.x] = cache[0];

     // une reduction final, sur le tableau des operations
       __syncthreads();
       unsigned int j = blockDim.x/2;
        while(i!=0)
        {
           if(tid < j)
           {   dev_resOperations[tid]=umin( dev_resOperations[tid], dev_resOperations[tid + j] );}
           __syncthreads();
           j/=2;
        }

}
