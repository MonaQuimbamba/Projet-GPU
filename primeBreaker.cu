#include "hip/hip_runtime.h"
#include "primeBreaker.hpp"


/** \brief
      je suis la focntion qui verifie la primalité d'un numero ,
      exemple d'éxecution : chaque thread verifie un numero inférieu à N peut diviser N
      si oui on ajout 0 dans le tableu de la memoire partage sinon on ajoute 1 , et au final on fait
      une reduction pour verifier si y'a des 0 dans la memoire partagée et on retourne un tableu de la taille
      d'un block avec que des 0 et 1 et dans CPU on verifie si ya des 0 c-à-d qu'il n'est pas premier sinon oui
*/
__global__ void isPrimeGPU(uint64_t *const dev_N, unsigned int  *const isPrime, uint64_t const N, size_t const taille)
{
      int global_t_id = threadIdx.x +  blockIdx.x*blockDim.x;
      int t_id = threadIdx.x;
      extern __shared__ unsigned int cache[];
      while( global_t_id < taille)
      {
          	cache[t_id]=  ((N%dev_N[global_t_id])==0 ) ? 0 : 1;
            global_t_id+=blockDim.x * gridDim.x;
      }
      __syncthreads();
        /*unsigned int i = blockDim.x/2;
      	while(i!=0)
      	{
      		 if(t_id < i)
      		 {
      			 cache[t_id]=umin( cache[t_id], cache[t_id + i] );
      		 }
      		 __syncthreads();
      		 i/=2;
      	}*/
    	unsigned int i = 1;
    	while ( i < blockDim.x )
    	{
    		int id = 2 * i * t_id;
    		if ( id < blockDim.x )
    		{
    			cache[id] = umin(cache[id],cache[id+i]);
    		}
    		__syncthreads();
    		i *= 2;
    	}
       if(threadIdx.x==0)  isPrime[blockIdx.x] = cache[0];


}
