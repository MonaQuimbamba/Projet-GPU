
#include <iostream>
#include <cstdlib>
#include <cstdint>
#include "utils/chronoCPU.hpp"
#include "utils/chronoGPU.hpp"
#include "primeBreakerCPU.hpp"
#include "primeBreaker.hpp"
#include "utils/common.hpp"

using namespace std;

void printUsage( const char *prg )
{
	cerr	<< "Usage: " << prg << endl
			<< " \t N "
			<< endl << endl;
	exit( EXIT_FAILURE );
}

string printPrimes(std::vector<uint64_t> primeNumbers)
{
        string res =  "Nombres premiers : \n " ;

        for(int i =0 ; i < primeNumbers.size() ; i++)
        {
           res += "[" + std::to_string(primeNumbers.at(i)) + "]";
        }
    return res;
}

string printFactuers(vector<cell> facteurs )
{
    string res = "Les Facteurs premiers :  \n ";
    for(int i = 0 ; i < facteurs.size(); i++)
    {
			string cell = to_string(facteurs.at(i).base)+"^"+to_string(facteurs.at(i).expo);
    	res+= (i==facteurs.size()-1) ? ""+cell : cell+"*" ;
    }
    return res;
}

int main( int argc, char **argv )
{


	uint64_t N =200;

	cout << "============================================"	<< endl;
	cout << "         Sequential version on CPU          " 	<< endl;
	cout << "============================================"	<< endl << endl;


	cout << " Partie CPU sur le nombre  " + to_string(N)<< endl;
	ChronoCPU chrCPU;
	chrCPU.start();
	bool isPrime = isPrimeCPU(N);
	chrCPU.stop();
	const float timeComputeCPUIsPrime = chrCPU.elapsedTime();
	cout << "Temps du test de primalite : "	<< timeComputeCPUIsPrime << " ms" << endl;
	cout << " Est Premier ? " << isPrime << endl;

	cout << " Recherche des nombres premiers sur CPU " << endl;
	chrCPU.start();
	std::vector<uint64_t> primesNumbers = searchPrimesCPU(N);
	chrCPU.stop();
	const float timeComputeCPUSearchPrime = chrCPU.elapsedTime();
  //cout << printPrimes(primesNumbers) << endl; //afficher les nombres premiers
    cout << "Temps de recherche : "	<< timeComputeCPUSearchPrime << " ms" << endl;

	cout << " Factorisation en nombre premier  sur CPU " << endl;
	chrCPU.start();
  vector<cell> facteurs(0);
  factoCPU(N,&facteurs);

	chrCPU.stop();
	const float timeComputeCPUFact = chrCPU.elapsedTime();
	cout << "Temps de factorisation en nombre premier : "	<< timeComputeCPUFact << " ms" << endl;
	cout << " Factorisation CPU : " << printFactuers(facteurs)<<endl ; // ajouter une focntion pour afficher la factorisation de cette façon 2133=1 ∗ 3^3 ∗ 79^1


	cout << "============================================"	<< endl;
	cout << "          Parallel versions on GPU           "	<< endl;
	cout << "============================================"	<< endl << endl;

	cout << " Partie GPU sur le nombre : " + to_string(N)<< endl;
  unsigned int isPrimeGPU=1;
  uint64_t *dev_N;
	uint64_t  *tab;
	tab = (uint64_t*)malloc( N*sizeof(uint64_t) );
	for (uint64_t i= 0;i < N; i++)
	{
				if(i==0 || i==1)
				{
					tab[i]=2;
				}
				else{
					tab[i]=i;
				}
  }

  HANDLE_ERROR(hipMalloc( (void**)&dev_N,  N*sizeof(uint64_t) ));
  HANDLE_ERROR(hipMemcpy(dev_N,tab, N * sizeof(uint64_t), hipMemcpyHostToDevice ));
  float timeComputeGPUIsPrime = launchKernelIsPrimeGPU<0>(dev_N,isPrimeGPU,N);
  cout << "Temps du test de primalite : "	<< timeComputeGPUIsPrime << " ms" << endl;
  cout << " Est Premier ? : " << N << " -> "<< isPrimeGPU <<endl;

	free( tab );
	HANDLE_ERROR(hipFree( dev_N ));



/*
	cout << " Recherche des nombres premiers sur GPU " << endl;
	float timeComputeGPUSearch = searchPrimesGPU<0>( N);
	cout << "Temps de recherche : "	<< timeComputeGPUSearch << " ms" << endl;

	cout << " Factorisation en nombre premier  sur GPU " << endl;
	float timeComputeGPUFact = factoGPU<0>( N);
	cout << "Temps de factorisation en nombre premier : "	<< timeComputeGPUFact << " ms" << endl;
	cout << " Factorisation GPU : " ; // ajouter une focntion pour afficher la factorisation de cette façon 2133=1 ∗ 3^3 ∗ 79^1

	// librerer la memoire du device ici
*/


	return EXIT_SUCCESS;
}
