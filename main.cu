
#include <iostream>
#include <cstdlib>
#include <cstdint>
#include "utils/chronoCPU.hpp"
#include "utils/chronoGPU.hpp"
#include "primeBreakerCPU.hpp"
#include "primeBreaker.hpp"

using namespace std;

void printUsage( const char *prg )
{
	cerr	<< "Usage: " << prg << endl
			<< " \t N "
			<< endl << endl;
	exit( EXIT_FAILURE );
}

string printPrimes(std::vector<uint64_t> primeNumbers)
{
        string res =  "Nombres premiers : \n " ;

        for(int i =0 ; i < primeNumbers.size() ; i++)
        {
           res += "[" + std::to_string(primeNumbers.at(i)) + "]";
        }
    return res;
}

string printFactuers(vector<uint64_t> facteurs )
{
    string res = "Les Facteurs premiers :  \n ";
    for(int i = 0 ; i < facteurs.size(); i++)
    {
        res+=  (i==facteurs.size()-1) ? ""+to_string(facteurs.at(i)) : ""+to_string(facteurs.at(i))+"*" ;
    }
    return res;
}

int main( int argc, char **argv )
{


	uint64_t N =33;
    /*
	if(N==0)
	{
		printUsage( argv[0] );
	}

	if( argc==1)
	{
			if ( sscanf( argv[1],"%" SCNu64,&N ) != 1 )
			{
						printUsage( argv[0] );
			}

	}

	cout << "%lu64" , N ;
*/


	cout << "============================================"	<< endl;
	cout << "         Sequential version on CPU          " 	<< endl;
	cout << "============================================"	<< endl << endl;


	cout << " Partie CPU sur le nombre  " + to_string(N)<< endl;
	ChronoCPU chrCPU;
	chrCPU.start();
	bool isPrime = isPrimeCPU(N);
	chrCPU.stop();
	const float timeComputeCPUIsPrime = chrCPU.elapsedTime();
	cout << "Temps du test de primalite : "	<< timeComputeCPUIsPrime << " ms" << endl;
	cout << " Est Premier ? " << isPrime << endl;

	cout << " Recherche des nombres premiers sur CPU " << endl;
	chrCPU.start();
	std::vector<uint64_t> primesNumbers = searchPrimesCPU(N);
	chrCPU.stop();
	const float timeComputeCPUSearchPrime = chrCPU.elapsedTime();
    cout << printPrimes(primesNumbers) << endl;
    cout << "Temps de recherche : "	<< timeComputeCPUSearchPrime << " ms" << endl;

	cout << " Factorisation en nombre premier  sur CPU " << endl;
	chrCPU.start();
    vector<uint64_t> facteurs(0);
     factoCPU(N,&facteurs);

	chrCPU.stop();
	const float timeComputeCPUFact = chrCPU.elapsedTime();
	cout << "Temps de factorisation en nombre premier : "	<< timeComputeCPUFact << " ms" << endl;
	cout << " Factorisation CPU : " << printFactuers(facteurs)<<endl ; // ajouter une focntion pour afficher la factorisation de cette façon 2133=1 ∗ 3^3 ∗ 79^1



	cout << "============================================"	<< endl;
	cout << "          Parallel versions on GPU           "	<< endl;
	cout << "============================================"	<< endl << endl;

	cout << " Partie GPU sur le nombre " + to_string(N)<< endl;
    unsigned int isPrimeGPU;
    uint64_t *dev_N;
		uint64_t  *tab;
		tab = (uint64_t*)malloc( N*sizeof(uint64_t) );
		for (long int i= 0;i < N; i++)
		{
					if(i==0 || i==1)
					{
						tab[i]=2;
					}
					else{
						tab[i]=i;
					}
    }
    hipMalloc( (void**)&dev_N, N*sizeof(uint64_t) );
    hipMemcpy(dev_N,tab, N * sizeof(uint64_t), hipMemcpyHostToDevice );
    float timeComputeGPUIsPrime = launchKernelIsPrimeGPU<0>(tab,N,isPrimeGPU);
	cout << "Temps du test de primalite : "	<< timeComputeGPUIsPrime << " ms" << endl;
	cout << " Est Premier ? : " ; // afficher like  2133 −> 0
/*
	cout << " Recherche des nombres premiers sur GPU " << endl;
	float timeComputeGPUSearch = searchPrimesGPU<0>( N);
	cout << "Temps de recherche : "	<< timeComputeGPUSearch << " ms" << endl;

	cout << " Factorisation en nombre premier  sur GPU " << endl;
	float timeComputeGPUFact = factoGPU<0>( N);
	cout << "Temps de factorisation en nombre premier : "	<< timeComputeGPUFact << " ms" << endl;
	cout << " Factorisation GPU : " ; // ajouter une focntion pour afficher la factorisation de cette façon 2133=1 ∗ 3^3 ∗ 79^1

	// librerer la memoire du device ici
*/


	return EXIT_SUCCESS;
}
