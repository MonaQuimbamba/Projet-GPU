#include "hip/hip_runtime.h"
#include "TestPrimeBreakerGPU.hpp"

/**	\brief	Je suis une fonction qui lance les tests unitaires pour les 
 * 		calculs a effectuer sur le GPU.
 */
void launchUnitTestGPU(){
    cout << "============================================"	<< endl;
    cout << "         Lancement des tests unitaires.     " 	<< endl;
    cout << "============================================"	<< endl << endl;

    testIfNonPrimeIsNotAssertedWithAIntegerPrimeNumberOnGPU();
    testIfPrimeIsAssertedWithAIntegerPrimeNumberOnGPU();
/*
    testIfPrimeIsAssertedWithALargeUint64PrimeNumberOnGPU();
    testIfNonPrimeIsNotAssertedWithALargeUint64PrimeNumberOnGPU();
    TestIfPrimesBetween0and100AreSuccessfullyRetrieved();
*/
    cout << "============================================"	<< endl;
    cout << "    Tests unitaires éffectués avec succès.   " 	<< endl;
    cout << "============================================"	<< endl << endl;
}


/**
 * \brief   Tester si un nombre non premier assez grand (tenant sur un UINT32_T) n'est pas reconnu
 *          comme un nombre premier par notre fonction.
 */
void testIfNonPrimeIsNotAssertedWithAIntegerPrimeNumberOnGPU(){
    std::cout << "Tester si un nombre non premier assez large tenant sur un UINT32_T n'est pas reconnu comme tel." << std::endl;
    
    int taille = (sqrt(UINT32_T_PRIME-1)+1)-2; // [[2;sqrt(N)+1]]
    uint64_t *possibles_diviseurs = (uint64_t *)malloc(sizeof(uint64_t) * taille);
    unsigned int *resOperations = (unsigned int *)malloc(sizeof(unsigned int *) * taille);
    memset(resOperations, '\0', taille);
    for(int i = 0,j = 2; i < taille; possibles_diviseurs[i] = j,i++,j++);
    
    
    uint64_t *dev_possibles_diviseurs = NULL;
    unsigned int *dev_resOperations = NULL;
    
    hipMalloc(&dev_possibles_diviseurs, sizeof(uint64_t) * taille);
    hipMalloc(&dev_resOperations, sizeof(unsigned int) * taille);
    hipMemcpy(dev_possibles_diviseurs, possibles_diviseurs, sizeof(uint64_t) * taille, hipMemcpyHostToDevice);
    hipMemcpy(dev_resOperations, resOperations, sizeof(unsigned int) * taille, hipMemcpyHostToDevice);

    isPrimeGPU<<<GRIDDIM(taille),BLOCKDIM>>>(	
	dev_possibles_diviseurs,
	dev_resOperations,
	UINT32_T_PRIME-1,
	taille
	);

    hipMemcpy(resOperations, dev_resOperations, sizeof(unsigned int) * taille, hipMemcpyDeviceToHost);
    
    hipFree(dev_possibles_diviseurs);
    hipFree(dev_resOperations);
    free(possibles_diviseurs);
    free(resOperations);
    
    mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT32_T_PRIME-1\n\ttaille)\n",
	resOperations[0] == 0,
	"Le nombre non premier a été reconnu comme un nombre premier.\n");


    std::cout << "Le nombre non premier n'a pas été reconnu : succès." << std::endl << std::endl;
}

/**
 * \brief   Tester si un nombre premier assez grand (tenant sur un UINT32_T) est reconnu comme tel par
 *          notre fonction.
 */
void testIfPrimeIsAssertedWithAIntegerPrimeNumberOnGPU(){
    std::cout << "Tester si un nombre premier assez large tenant sur un UINT32_T est reconnu comme tel." << std::endl;

    int taille = (sqrt(UINT32_T_PRIME)+1)-2; // [[2;sqrt(N)+1]]
    uint64_t *possibles_diviseurs = (uint64_t *)malloc(sizeof(uint64_t) * taille);
    unsigned int *resOperations = (unsigned int *)malloc(sizeof(unsigned int *) * taille);
    memset(resOperations, '\0', taille);
    for(int i = 0, j = 2; i < taille; possibles_diviseurs[i] = j,i++,j++);
    
    uint64_t *dev_possibles_diviseurs = NULL;
    unsigned int *dev_resOperations = NULL;
    
    hipMalloc(&dev_possibles_diviseurs, sizeof(uint64_t) * taille);
    hipMalloc(&dev_resOperations, sizeof(unsigned int) * taille);
    hipMemcpy(dev_possibles_diviseurs, possibles_diviseurs, sizeof(uint64_t) * taille, hipMemcpyHostToDevice);
    hipMemcpy(dev_resOperations, resOperations, sizeof(unsigned int) * taille, hipMemcpyHostToDevice);

    isPrimeGPU<<<GRIDDIM(taille),BLOCKDIM>>>(	
	dev_possibles_diviseurs,
	dev_resOperations,
	UINT32_T_PRIME,
	taille
	);

    hipMemcpy(resOperations, dev_resOperations, sizeof(unsigned int) * taille, hipMemcpyDeviceToHost);
    
    hipFree(dev_possibles_diviseurs);
    hipFree(dev_resOperations);
    free(possibles_diviseurs);
    free(resOperations);
    
    mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT32_T_PRIME\n\ttaille)\n",
	resOperations[0] == 1,
	"Le nombre premier n'a pas été reconnu comme tel.\n");


    std::cout << "Le nombre premier a été reconnu : succès." << std::endl << std::endl;
}

/**
 * \brief   Tester si un nombre premier assez grand (tenant sur un UINT64_T) est reconnu comme tel par
 *          notre fonction.
 */
void testIfPrimeIsAssertedWithALargeUint64PrimeNumberOnGPU(){
    std::cout << "Tester si un nombre premier tenant sur un UINT64_T est reconnu comme tel." << std::endl;

    std::cout << "Le nombre premier a été reconnu : succès." << std::endl << std::endl;

}

/**
 * \brief   Tester si un nombre non premier assez grand (tenant sur un UINT64_T) n'est  pas reconnu
 *          comme un nombre premier par notre fonction.
 */
void testIfNonPrimeIsNotAssertedWithALargeUint64PrimeNumberOnGPU(){
    std::cout << "Tester si un nombre non premier tenant sur un UINT64_T n'est pas reconnu comme tel." << std::endl;

    std::cout << "Le nombre non premier n'a pas été reconnu : succès " << std::endl
    << std::endl;

}
