#include "hip/hip_runtime.h"
#include "TestPrimeBreakerGPU.hpp"

/**	\brief	Je suis une fonction qui lance les tests unitaires pour les 
 * 		calculs a effectuer sur le GPU.
 */
void launchUnitTestGPU(){
    cout << "============================================"	<< endl;
    cout << "         Lancement des tests unitaires.     " 	<< endl;
    cout << "============================================"	<< endl << endl;

    testIfNonPrimeIsNotAssertedWithAIntegerPrimeNumberOnGPU();
    testIfPrimeIsAssertedWithAIntegerPrimeNumberOnGPU();

    testIfPrimeIsAssertedWithALargeUint64PrimeNumberOnGPU();
    testIfNonPrimeIsNotAssertedWithALargeUint64PrimeNumberOnGPU();
/*
    TestIfPrimesBetween0and100AreSuccessfullyRetrieved();
*/
    cout << "============================================"	<< endl;
    cout << "    Tests unitaires éffectués avec succès.   " 	<< endl;
    cout << "============================================"	<< endl << endl;
}


/**
 * \brief   Tester si un nombre non premier assez grand (tenant sur un UINT32_T) n'est pas reconnu
 *          comme un nombre premier par notre fonction.
 */
void testIfNonPrimeIsNotAssertedWithAIntegerPrimeNumberOnGPU(){
    	std::cout << "Tester si un nombre non premier assez large tenant sur un UINT32_T n'est pas reconnu comme tel." << std::endl;

	uint64_t N = UINT32_T_PRIME-1;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM(BLOCKDIM)>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT32_T_PRIME-1\n\ttaille)\n",
		res_operations[0] == 0,
		"Le nombre non premier a été reconnu comme un nombre premier.\n");

	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);

    	std::cout << "Le nombre non premier n'a pas été reconnu : succès." << std::endl << std::endl;
}

/**
 * \brief   Tester si un nombre premier assez grand (tenant sur un UINT32_T) est reconnu comme tel par
 *          notre fonction.
 */
void testIfPrimeIsAssertedWithAIntegerPrimeNumberOnGPU(){
    	std::cout << "Tester si un nombre premier assez large tenant sur un UINT32_T est reconnu comme tel." << std::endl;
    
	uint64_t N = UINT32_T_PRIME;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM(BLOCKDIM)>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT32_T_PRIME\n\ttaille)\n",
		res_operations[0] == 1,
		"Le nombre premier n'a pas été reconnu comme tel.\n");
	
	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);

    	std::cout << "Le nombre premier a été reconnu : succès." << std::endl << std::endl;
}

/**
 * \brief   Tester si un nombre premier assez grand (tenant sur un UINT64_T) est reconnu comme tel par
 *          notre fonction.
 */
void testIfPrimeIsAssertedWithALargeUint64PrimeNumberOnGPU(){
    std::cout << "Tester si un nombre premier tenant sur un UINT64_T est reconnu comme tel." << std::endl;
	
    	uint64_t N = UINT64_T_PRIME;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM(BLOCKDIM)>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

    	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT64_T_PRIME\n\ttaille)\n",
		res_operations[0] == 1,
		"Le nombre premier n'a pas été reconnu comme tel.\n");
	
	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);

    std::cout << "Le nombre premier a été reconnu : succès." << std::endl << std::endl;

}

/**
 * \brief   Tester si un nombre non premier assez grand (tenant sur un UINT64_T) n'est  pas reconnu
 *          comme un nombre premier par notre fonction.
 */
void testIfNonPrimeIsNotAssertedWithALargeUint64PrimeNumberOnGPU(){
    std::cout << "Tester si un nombre non premier tenant sur un UINT64_T n'est pas reconnu comme tel." << std::endl;
	
    	uint64_t N = UINT64_T_PRIME-1;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM(BLOCKDIM)>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

    	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT64_T_PRIME-1\n\ttaille)\n",
		res_operations[0] == 0,
		"Le nombre non premier a été reconnu comme un nombre premier.\n");

	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);


    std::cout << "Le nombre non premier n'a pas été reconnu : succès " << std::endl
    << std::endl;
}
