#include "hip/hip_runtime.h"
#include "TestPrimeBreakerGPU.hpp"

/**	\brief	Je suis une fonction qui lance les tests unitaires pour les 
 * 		calculs a effectuer sur le GPU.
 */
void launchUnitTestGPU(){
    cout << "============================================"	<< endl;
    cout << "         Lancement des tests unitaires.     " 	<< endl;
    cout << "============================================"	<< endl << endl;

    testIfNonPrimeIsNotAssertedWithAIntegerPrimeNumberOnGPU();
    testIfPrimeIsAssertedWithAIntegerPrimeNumberOnGPU();
/*
    testIfPrimeIsAssertedWithALargeUint64PrimeNumberOnGPU();
    testIfNonPrimeIsNotAssertedWithALargeUint64PrimeNumberOnGPU();
    TestIfPrimesBetween0and100AreSuccessfullyRetrieved();
*/
    cout << "============================================"	<< endl;
    cout << "    Tests unitaires éffectués avec succès.   " 	<< endl;
    cout << "============================================"	<< endl << endl;
}


/**
 * \brief   Tester si un nombre non premier assez grand (tenant sur un UINT32_T) n'est pas reconnu
 *          comme un nombre premier par notre fonction.
 */
void testIfNonPrimeIsNotAssertedWithAIntegerPrimeNumberOnGPU(){
    	std::cout << "Tester si un nombre non premier assez large tenant sur un UINT32_T n'est pas reconnu comme tel." << std::endl;

	unsigned int N = UINT32_T_PRIME-1;
	unsigned int sqrtN = sqrt(N) + 1;
	unsigned int nombresDePossiblesPremiers = N-2;

	unsigned int *possibles_premiers = (unsigned int*)malloc(sizeof(unsigned int) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	unsigned int *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(unsigned int) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(unsigned int) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM(BLOCKDIM)>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

	for (int i = 0; i < GRIDDIM(sqrtN); i++){
		res_operations[i] == 0 ? res_operations[0] = 0 : 0;
	}
    
    	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT32_T_PRIME-1\n\ttaille)\n",
		res_operations[0] == 0,
		"Le nombre non premier a été reconnu comme un nombre premier.\n");

	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);

    	std::cout << "Le nombre non premier n'a pas été reconnu : succès." << std::endl << std::endl;
}

/**
 * \brief   Tester si un nombre premier assez grand (tenant sur un UINT32_T) est reconnu comme tel par
 *          notre fonction.
 */
void testIfPrimeIsAssertedWithAIntegerPrimeNumberOnGPU(){
    	std::cout << "Tester si un nombre premier assez large tenant sur un UINT32_T est reconnu comme tel." << std::endl;
    
	unsigned int N = UINT32_T_PRIME;
	unsigned int sqrtN = sqrt(N) + 1;
	unsigned int nombresDePossiblesPremiers = N-2;

	unsigned int *possibles_premiers = (unsigned int*)malloc(sizeof(unsigned int) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	unsigned int *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(unsigned int) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(unsigned int) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM(BLOCKDIM)>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

	for (int i = 0; i < GRIDDIM(sqrtN); i++){
		res_operations[i] == 0 ? res_operations[0] = 0 : 0;
	}
    
    	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT32_T_PRIME\n\ttaille)\n",
		res_operations[0] == 1,
		"Le nombre premier n'a pas été reconnu comme tel.\n");
	
	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);

    	std::cout << "Le nombre premier a été reconnu : succès." << std::endl << std::endl;
}

/**
 * \brief   Tester si un nombre premier assez grand (tenant sur un UINT64_T) est reconnu comme tel par
 *          notre fonction.
 */
void testIfPrimeIsAssertedWithALargeUint64PrimeNumberOnGPU(){
    std::cout << "Tester si un nombre premier tenant sur un UINT64_T est reconnu comme tel." << std::endl;

    std::cout << "Le nombre premier a été reconnu : succès." << std::endl << std::endl;

}

/**
 * \brief   Tester si un nombre non premier assez grand (tenant sur un UINT64_T) n'est  pas reconnu
 *          comme un nombre premier par notre fonction.
 */
void testIfNonPrimeIsNotAssertedWithALargeUint64PrimeNumberOnGPU(){
    std::cout << "Tester si un nombre non premier tenant sur un UINT64_T n'est pas reconnu comme tel." << std::endl;

    std::cout << "Le nombre non premier n'a pas été reconnu : succès " << std::endl
    << std::endl;

}
