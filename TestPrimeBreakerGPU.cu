#include "hip/hip_runtime.h"
#include "TestPrimeBreakerGPU.hpp"

/**	\brief	Je suis une fonction qui lance les tests unitaires pour les 
 * 		calculs a effectuer sur le GPU.
 */
void launchUnitTestGPU(){
    cout << "============================================"	<< endl;
    cout << "         Lancement des tests unitaires.     " 	<< endl;
    cout << "============================================"	<< endl << endl;

    testIfNonPrimeIsNotAssertedWithAIntegerPrimeNumberOnGPU();
    testIfPrimeIsAssertedWithAIntegerPrimeNumberOnGPU();
    testIfPrimeIsAssertedWithALargeUint64PrimeNumberOnGPU();
    testIfNonPrimeIsNotAssertedWithALargeUint64PrimeNumberOnGPU();
    testIfPrimesBetween0and100AreComputedOnGPU();
    //testIfNumberIsFactorized();
    cout << "============================================"	<< endl;
    cout << "    Tests unitaires éffectués avec succès.   " 	<< endl;
    cout << "============================================"	<< endl << endl;
}


/**
 * \brief   Tester si un nombre non premier assez grand (tenant sur un UINT32_T) n'est pas reconnu
 *          comme un nombre premier par notre fonction.
 */
void testIfNonPrimeIsNotAssertedWithAIntegerPrimeNumberOnGPU(){
    	std::cout << "Tester si un nombre non premier assez large tenant sur un UINT32_T n'est pas reconnu comme tel." << std::endl;

	uint64_t N = UINT32_T_PRIME-1;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));

	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT32_T_PRIME-1\n\ttaille)\n",
		res_operations[0] == 0,
		"Le nombre non premier a été reconnu comme un nombre premier.\n");

	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);

    	std::cout << "Le nombre non premier n'a pas été reconnu : succès." << std::endl << std::endl;
}

/**
 * \brief   Tester si un nombre premier assez grand (tenant sur un UINT32_T) est reconnu comme tel par
 *          notre fonction.
 */
void testIfPrimeIsAssertedWithAIntegerPrimeNumberOnGPU(){
    	std::cout << "Tester si un nombre premier assez large tenant sur un UINT32_T est reconnu comme tel." << std::endl;
    
	uint64_t N = UINT32_T_PRIME;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT32_T_PRIME\n\ttaille)\n",
		res_operations[0] == 1,
		"Le nombre premier n'a pas été reconnu comme tel.\n");
	
	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);

    	std::cout << "Le nombre premier a été reconnu : succès." << std::endl << std::endl;
}

/**
 * \brief   Tester si un nombre premier assez grand (tenant sur un UINT64_T) est reconnu comme tel par
 *          notre fonction.
 */
void testIfPrimeIsAssertedWithALargeUint64PrimeNumberOnGPU(){
    std::cout << "Tester si un nombre premier tenant sur un UINT64_T est reconnu comme tel." << std::endl;
	
    	uint64_t N = UINT64_T_PRIME;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

    	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT64_T_PRIME\n\ttaille)\n",
		res_operations[0] == 1,
		"Le nombre premier n'a pas été reconnu comme tel.\n");
	
	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);

    std::cout << "Le nombre premier a été reconnu : succès." << std::endl << std::endl;

}

/**
 * \brief   Tester si un nombre non premier assez grand (tenant sur un UINT64_T) n'est  pas reconnu
 *          comme un nombre premier par notre fonction.
 */
void testIfNonPrimeIsNotAssertedWithALargeUint64PrimeNumberOnGPU(){
    std::cout << "Tester si un nombre non premier tenant sur un UINT64_T n'est pas reconnu comme tel." << std::endl;
	
    	uint64_t N = UINT64_T_PRIME-1;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

    	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT64_T_PRIME-1\n\ttaille)\n",
		res_operations[0] == 0,
		"Le nombre non premier a été reconnu comme un nombre premier.\n");

	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);


    std::cout << "Le nombre non premier n'a pas été reconnu : succès " << std::endl
    << std::endl;
}

void testIfPrimesBetween0and100AreComputedOnGPU(){
    std::cout << "Tester la récupération des nombres premiers entre 0 et 100." << std::endl;

	vector<uint64_t> controlPrimeSet = getPrimesFrom0to100FromControlPrimeSetFile();

	uint64_t borne_sup = 100;
	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); possibles_premiers[i] = i+2, i++);
	uint64_t *square_roots = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); square_roots[i] = sqrt(i+2), i++);
	uint64_t *premiers = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); premiers[i] = 0, i++);

	if (VERBOSE) {
		cout << "Afficher les données initialisées " << endl;
		cout << "possibles premiers ";
		for (int i = 0; i < (borne_sup-2); i++){
			cout << "[" << possibles_premiers[i] << "]";
		}
		cout << endl << endl;
		
		cout << "square_roots ";
		for (int i = 0; i < (borne_sup-2); i++){
			cout << "[" << square_roots[i] << "]";
		}
		cout << endl << endl;
		
		cout << "premiers ";
		for (int i = 0; i < (borne_sup-2); i++){
			cout << "[" << premiers[i] << "]";
		}
		cout << endl << endl;

		cout << "Fin affichage des données initialisées" << endl;
	}


	uint64_t *dev_possibles_premiers;
	uint64_t *dev_square_roots;
	uint64_t *dev_premiers;

	hipMalloc((void**)&dev_possibles_premiers,sizeof(uint64_t)*(borne_sup-2));
	hipMalloc((void**)&dev_square_roots,sizeof(uint64_t)*(borne_sup-2));
	hipMalloc((void**)&dev_premiers,sizeof(uint64_t)*(borne_sup-2));
	
	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	hipMemcpy(dev_square_roots, square_roots, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	hipMemcpy(dev_premiers, premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);

	searchPrimeGPU<<<GRIDDIM(borne_sup-2),BLOCKDIM,SIZEMEM>>>(
			dev_possibles_premiers, 
			dev_square_roots, 
			borne_sup, 
			dev_premiers);

	hipMemcpy(premiers, dev_premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyDeviceToHost);

	if (VERBOSE) {
		cout << "Affichage du tableau premiers après calcul GPU" << endl;
		for (int i = 0; i < borne_sup-2; i++){
			cout << "[" << i+2 << "," << premiers[i] << "]";
		}
		cout << endl << "Fin Affichage" << endl << endl; 
	}

    // Début Assertions

	int nombresDePremiers = 0;
	for(int i = 0; i < (borne_sup-2); i++){
		if (premiers[i] != 0)
			nombresDePremiers++;
	}
	uint64_t premiers_packed[nombresDePremiers];
	for (int i = 0; i < nombresDePremiers; i++){
		int j = 0;

		while (premiers[j] == 0 && j < (borne_sup-2))
			j++;
		premiers_packed[i] = (j+2);
		premiers[j] = 0;
	}

	if (VERBOSE)

        {
		printf("Liste de nombres premiers récupérés du GPU : \n");
		for (int i = 0; i < nombresDePremiers; i++){
			printf("[%lld]", premiers_packed[i]);
		}

		printf("\n\nListe des nombres premiers récupérés du témoin : \n");
		for (int i = 0; i < controlPrimeSet.size(); i++){
			printf("[%lld]", controlPrimeSet.at(i));
		}
	}

    mAssert("controlPrimeSet.size() == nombresDePremiers",
            controlPrimeSet.size() == nombresDePremiers,
            string("La fonction ne renvoit pas le même nombre de nombres premiers que dans le groupe de controle.\n")
            + string("controlPrimeSet.size() = ") + std::to_string(controlPrimeSet.size()) +
            string("\nprimesNumberFrom0to100.size() = ") + std::to_string(nombresDePremiers)
            + string("\n")
    );

    for (int i = 0; i < controlPrimeSet.size(); i++){
        mAssert("controlPrimeSet.at(i) == primesNumberFrom0to100.at(1)",
                controlPrimeSet.at(i) == premiers_packed[i],
                ("On ne retrouve pas le " + std::to_string(i) + "ème nombre premier.")
                );
    }

    std::cout << "On retrouve bien tout les nombres premiers compris dans l'interval : Succès." << std::endl;


	// appeler le tes Fact  en profitant  de la liste des premiers 
       testIfNumberIsFactorized(premiers_packed,nombresDePremiers);

}



void  testIfNumberIsFactorized(uint64_t *primes,int taille)
{


        uint64_t N=100;
	cell cinq;
	cinq.base=5;
	cinq.expo=2;
	cell deux;
	deux.base=2;
	deux.expo=2;
        cell  *facteurs=(cell*)malloc(sizeof(cell)*taille);
	// on remplie le tableua de cell avec l'ensemble des nombres premiers tous avec un exposant de 0 
	for(int i =0 ; i<taille; facteurs[i].base=primes[i],facteurs[i].expo=0,i++);

	uint64_t *dev_primes;
	cell *dev_facteurs;
	
	hipMalloc((void**)&dev_primes,sizeof(uint64_t)*taille);
        hipMalloc((void**)&dev_facteurs,sizeof(cell)*taille);
       
	hipMemcpy(dev_primes,primes,sizeof(uint64_t)*taille,hipMemcpyHostToDevice);
	hipMemcpy(dev_facteurs,facteurs,sizeof(cell)*taille,hipMemcpyHostToDevice);
     
      uint64_t *val;
      
     hipMallocManaged(&val,sizeof(uint64_t));
     *val=N;
     while(*val!=1)
      {
           N=*val;
          // printf("val: {%lld} ",N);
           factGPU<<<GRIDDIM(taille),BLOCKDIM>>>(N,dev_primes,dev_facteurs,taille,val);
           hipDeviceSynchronize();
      } 
	hipMemcpy(facteurs,dev_facteurs,sizeof(cell)*taille,hipMemcpyDeviceToHost);

	cell cinq_;
        cell deux_;
        cinq_.base=5;
      
        deux_.base=2;
        for(int i=0 ; i< taille ; i++)
	{

	cout << " :  "<< facteurs[i].base <<"^"<<facteurs[i].expo<<endl<<endl;

		if(facteurs[i].base==cinq_.base)
		{cinq_.expo=facteurs[i].expo;}

		if(facteurs[i].base==deux_.base)
		{deux_.expo=facteurs[i].expo;}
	}


      	 mAssert("l'exposant de la base cinq  ",
               cinq.expo==cinq_.expo,
              "la base cinq n'as pas le bon exposant  ");

	mAssert("l'exposant de la base deux ",
		deux.expo==deux_.expo,
		"la base deux n'a pas le bon exposant");

	cout<<" La factorisation a bien focntionnée : Succès "<<endl<<endl;
	
}
	
		
