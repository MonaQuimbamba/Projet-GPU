#include "hip/hip_runtime.h"
#include "TestPrimeBreakerGPU.hpp"

/**	\brief	Je suis une fonction qui lance les tests unitaires pour les 
 * 		calculs a effectuer sur le GPU.
 */
void launchUnitTestGPU(){
    cout << "============================================"	<< endl;
    cout << "         Lancement des tests unitaires.     " 	<< endl;
    cout << "============================================"	<< endl << endl;

    testIfNonPrimeIsNotAssertedWithAIntegerPrimeNumberOnGPU();
    testIfPrimeIsAssertedWithAIntegerPrimeNumberOnGPU();
    testIfPrimeIsAssertedWithALargeUint64PrimeNumberOnGPU();
    testIfNonPrimeIsNotAssertedWithALargeUint64PrimeNumberOnGPU();
    testIfPrimesBetween0and100AreComputedOnGPU();

    cout << "============================================"	<< endl;
    cout << "    Tests unitaires éffectués avec succès.   " 	<< endl;
    cout << "============================================"	<< endl << endl;
}


/**
 * \brief   Tester si un nombre non premier assez grand (tenant sur un UINT32_T) n'est pas reconnu
 *          comme un nombre premier par notre fonction.
 */
void testIfNonPrimeIsNotAssertedWithAIntegerPrimeNumberOnGPU(){
    	std::cout << "Tester si un nombre non premier assez large tenant sur un UINT32_T n'est pas reconnu comme tel." << std::endl;

	uint64_t N = UINT32_T_PRIME-1;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT32_T_PRIME-1\n\ttaille)\n",
		res_operations[0] == 0,
		"Le nombre non premier a été reconnu comme un nombre premier.\n");

	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);

    	std::cout << "Le nombre non premier n'a pas été reconnu : succès." << std::endl << std::endl;
}

/**
 * \brief   Tester si un nombre premier assez grand (tenant sur un UINT32_T) est reconnu comme tel par
 *          notre fonction.
 */
void testIfPrimeIsAssertedWithAIntegerPrimeNumberOnGPU(){
    	std::cout << "Tester si un nombre premier assez large tenant sur un UINT32_T est reconnu comme tel." << std::endl;
    
	uint64_t N = UINT32_T_PRIME;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT32_T_PRIME\n\ttaille)\n",
		res_operations[0] == 1,
		"Le nombre premier n'a pas été reconnu comme tel.\n");
	
	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);

    	std::cout << "Le nombre premier a été reconnu : succès." << std::endl << std::endl;
}

/**
 * \brief   Tester si un nombre premier assez grand (tenant sur un UINT64_T) est reconnu comme tel par
 *          notre fonction.
 */
void testIfPrimeIsAssertedWithALargeUint64PrimeNumberOnGPU(){
    std::cout << "Tester si un nombre premier tenant sur un UINT64_T est reconnu comme tel." << std::endl;
	
    	uint64_t N = UINT64_T_PRIME;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

    	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT64_T_PRIME\n\ttaille)\n",
		res_operations[0] == 1,
		"Le nombre premier n'a pas été reconnu comme tel.\n");
	
	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);

    std::cout << "Le nombre premier a été reconnu : succès." << std::endl << std::endl;

}

/**
 * \brief   Tester si un nombre non premier assez grand (tenant sur un UINT64_T) n'est  pas reconnu
 *          comme un nombre premier par notre fonction.
 */
void testIfNonPrimeIsNotAssertedWithALargeUint64PrimeNumberOnGPU(){
    std::cout << "Tester si un nombre non premier tenant sur un UINT64_T n'est pas reconnu comme tel." << std::endl;
	
    	uint64_t N = UINT64_T_PRIME-1;
	uint64_t sqrtN = sqrt(N) + 1;
	uint64_t nombresDePossiblesPremiers = N-2;

	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t) * (nombresDePossiblesPremiers));
	for (int i = 0, j = 2.0; j < N; possibles_premiers[i] = j,i++,j++);
	unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));
	for (int i = 0; i < GRIDDIM(sqrtN); res_operations[i] = 1,i++);

	uint64_t *dev_possibles_premiers;
	hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers));
	unsigned int *dev_res_operations;
	hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));


	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN);
	hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);

    	mAssert("isPrimeGPU(\tdev_possibles_diviseurs\n\tdev_resOperations\n\tUINT64_T_PRIME-1\n\ttaille)\n",
		res_operations[0] == 0,
		"Le nombre non premier a été reconnu comme un nombre premier.\n");

	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);


    std::cout << "Le nombre non premier n'a pas été reconnu : succès " << std::endl
    << std::endl;
}

void testIfPrimesBetween0and100AreComputedOnGPU(){
    std::cout << "Tester la récupération des nombres premiers entre 0 et 100." << std::endl;

	vector<uint64_t> controlPrimeSet = getPrimesFrom0to100FromControlPrimeSetFile();

	uint64_t borne_sup = 100;
	uint64_t *possibles_premiers = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); possibles_premiers[i] = i+2, i++);
	uint64_t *square_roots = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); square_roots[i] = sqrt(i+2), i++);
	uint64_t *premiers = (uint64_t*)malloc(sizeof(uint64_t)*(borne_sup-2));
	for(int i = 0; i < (borne_sup-2); premiers[i] = 0, i++);

	if (VERBOSE) {
		cout << "Afficher les données initialisées " << endl;
		cout << "possibles premiers ";
		for (int i = 0; i < (borne_sup-2); i++){
			cout << "[" << possibles_premiers[i] << "]";
		}
		cout << endl << endl;
		
		cout << "square_roots ";
		for (int i = 0; i < (borne_sup-2); i++){
			cout << "[" << square_roots[i] << "]";
		}
		cout << endl << endl;
		
		cout << "premiers ";
		for (int i = 0; i < (borne_sup-2); i++){
			cout << "[" << premiers[i] << "]";
		}
		cout << endl << endl;

		cout << "Fin affichage des données initialisées" << endl;
	}


	uint64_t *dev_possibles_premiers;
	uint64_t *dev_square_roots;
	uint64_t *dev_premiers;

	hipMalloc((void**)&dev_possibles_premiers,sizeof(uint64_t)*(borne_sup-2));
	hipMalloc((void**)&dev_square_roots,sizeof(uint64_t)*(borne_sup-2));
	hipMalloc((void**)&dev_premiers,sizeof(uint64_t)*(borne_sup-2));
	
	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	hipMemcpy(dev_square_roots, square_roots, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);
	hipMemcpy(dev_premiers, premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyHostToDevice);

	searchPrimeGPU<<<GRIDDIM(borne_sup-2),BLOCKDIM,SIZEMEM>>>(
			dev_possibles_premiers, 
			dev_square_roots, 
			borne_sup, 
			dev_premiers);

	hipMemcpy(premiers, dev_premiers, sizeof(uint64_t)*(borne_sup-2), hipMemcpyDeviceToHost);

	if (VERBOSE) {
		cout << "Affichage du tableau premiers après calcul GPU" << endl;
		for (int i = 0; i < borne_sup-2; i++){
			cout << "[" << premiers[i] << "]";
		}
		cout << "Fin Affichage" << endl << endl; 
	}

    // Début Assertions

	int nombresDePremiers = 0;
	for(int i = 0; i < (borne_sup-2); i++){
		if (premiers[i] != 0)
			nombresDePremiers++;
	}
	uint64_t premiers_packed[nombresDePremiers];
	for (int i = 0; i < nombresDePremiers; i++){
		int j = 0;

		while (premiers[j] == 0 && j < (borne_sup-2))
			j++;
		premiers_packed[i] = premiers[j];
		premiers[j] = 0;
	}

    mAssert("controlPrimeSet.size() == nombresDePremiers",
            controlPrimeSet.size() == nombresDePremiers,
            string("La fonction ne renvoit pas le même nombre de nombres premiers que dans le groupe de controle.\n")
            + string("controlPrimeSet.size() = ") + std::to_string(controlPrimeSet.size()) +
            string("\nprimesNumberFrom0to100.size() = ") + std::to_string(nombresDePremiers)
            + string("\n")
    );

    int i = controlPrimeSet.size()-1;
    int j = 0;
    for (; i >= 0; i-- && j++){
        mAssert("controlPrimeSet.at(i) == primesNumberFrom0to100.at(1)",
                controlPrimeSet.at(i) == premiers_packed[j],
                ("On ne retrouve pas le " + std::to_string(i) + "ème nombre premier.")
                );
    }

    std::cout << "On retrouve bien tout les nombres premiers compris dans l'interval : Succès." << std::endl;
}
