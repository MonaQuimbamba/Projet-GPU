#include "hip/hip_runtime.h"
#include "sampling.h"

/** \brief  Je suis une foncton qui aggrège les mesures de
 *          nos tests de performances pour une création de
 *          graphes ultérieure avec GnuPlot.
 * @return boost::tuple<vector<float>,vector<int>> couple de (Tableau de Mesures de temps, Tableau de Logarithme en base 2 des échantillons).
 *
boost::tuple<vector<float>,vector<uint64_t>>
createPrimalityTestsDatas()
{
    vector<uint64_t> samples = generatePrimalityTestsSamples();
    vector<float> timeMeasurements = generatePrimalityTestsMeasurement(samples);

    for (uint64_t log2Samples = 4,
            i = 0;
            i < LOG2MAX_ISP-4;
            log2Samples++,
            i++){
        /// Après la mesure des échantillons, remplacer les
        /// échantillons dans le tableau d'échantillons par
        /// leur logarithme en base 2.
        samples.at(i) = log2Samples;
    }

    return boost::tuple<vector<float>, vector<uint64_t>>{timeMeasurements,samples};
}
*/

void generateDataFilesCPU(){
	cout << " Lancement des tests de performances et génération des fichiers de données " << endl;
    generateResearchOfPrimesDataFileCPU();
    generatePrimalityTestDataFileCPU();
    generatePrimeFactorisationDataFileCPU(); 
    	cout << " Fin des tests de performances, les fichiers des résultats sont dans data/" << endl << endl;
}

void generateResearchOfPrimesDataFileCPU(){
	cout << " Génération des données pour la recherche de nombres premiers sur le CPU " << endl;
    vector<uint64_t> limits = generateResearchOfPrimesLimits();
    vector<float> timeMeasurements = generateResearchOfPrimesMeasurement(limits);

    for (uint64_t log2Samples = 2,
                 i = 0;
         i < LOG2MAX_ROP-4;
         log2Samples++,
                 i++){
        /// Après la mesure des échantillons, remplacer les
        /// échantillons dans le tableau d'échantillons par
        /// leur logarithme en base 2.
        limits.at(i) = log2Samples;
    }

    // Créer un fichier
    ofstream datafile;
    datafile.open("data/researchOfPrimesCPU.dat",ios::out);
    if (datafile.bad()){
	cout << "Problème à l'ouverture du fichier" << endl;
    }else {
	for (int i =0; i < limits.size(); i++){
		datafile << limits.at(i) << '\t' << timeMeasurements.at(i) << '\n';
	}
    }
    	cout << " Fin Génération " << endl;
}

void generatePrimalityTestDataFileCPU(){
	cout << " Génération des données pour le test de primalité sur le CPU " << endl;
    vector<uint64_t> samples = generatePrimalityTestsSamples();
    vector<float> timeMeasurements = generatePrimalityTestsMeasurement(samples);

    for (uint64_t log2Samples = 2,
                 i = 0;
         i < LOG2MAX_ISP-4;
         log2Samples++,
                 i++){
        /// Après la mesure des échantillons, remplacer les
        /// échantillons dans le tableau d'échantillons par
        /// leur logarithme en base 2.
        samples.at(i) = log2Samples;
    }

    // Créer un fichier
    ofstream datafile;
    datafile.open("data/primalityTestsDatasCPU.dat",ios::out);
    if (datafile.bad()){
	cout << "Problème à l'ouverture du fichier" << endl;
    }else {
	for (int i =0; i < samples.size(); i++){
		datafile << samples.at(i) << '\t' << timeMeasurements.at(i) << '\n';
	}
    }
	cout << " Fin Génération. " << endl;

}

void generatePrimeFactorisationDataFileCPU(){
	cout << " Génération des données pour la factorisation sur le CPU " << endl;
    vector<uint64_t> samples = generatePrimeFactorisationSamples();
    vector<float> timeMeasurements = generatePrimeFactorisationMeasurement(samples);

    // Créer un fichier
    ofstream datafile;
    datafile.open("data/factorisationDatasCPU.dat",ios::out);
    if (datafile.bad()){
	cout << "Problème à l'ouverture du fichier" << endl;
    }else {
	for (int i =0; i < samples.size(); i++){
		datafile << samples.at(i) << '\t' << timeMeasurements.at(i) << '\n';
	}
    }
	cout << " Fin Génération. " << endl;
}

void generateDataFilesGPU(){
	cout << " Lancement des tests de performances et génération des fichiers de données " << endl;
    //generateResearchOfPrimesDataFileGPU();
    generatePrimalityTestDataFileGPU();
    //generatePrimeFactorisationDataFileGPU(); 
    	cout << " Fin des tests de performances, les fichiers des résultats sont dans data/" << endl << endl;
}

void generateResearchOfPrimesDataFileGPU(){
	cout << " Génération des données pour la recherche de nombres premiers sur le GPU " << endl;
    vector<uint64_t> limits = generateResearchOfPrimesLimits();
    vector<float> timeMeasurements = generateGPUResearchOfPrimesMeasurement(limits);

    for (uint64_t log2Samples = 2,
                 i = 0;
         i < LOG2MAX_ROP-4;
         log2Samples++,
                 i++){
        /// Après la mesure des échantillons, remplacer les
        /// échantillons dans le tableau d'échantillons par
        /// leur logarithme en base 2.
        limits.at(i) = log2Samples;
    }

    // Créer un fichier
    ofstream datafile;
    datafile.open("data/researchOfPrimesGPU.dat",ios::out);
    if (datafile.bad()){
	cout << "Problème à l'ouverture du fichier" << endl;
    }else {
	for (int i =0; i < limits.size(); i++){
		datafile << limits.at(i) << '\t' << timeMeasurements.at(i) << '\n';
	}
    }
    	cout << " Fin Génération " << endl;
}

void generatePrimalityTestDataFileGPU(){
	cout << " Génération des données pour le test de primalité sur le GPU " << endl;
    vector<uint64_t> samples = generatePrimalityTestsSamples();
    vector<float> timeMeasurements = generateGPUPrimalityTestsMeasurement(samples);

    for (uint64_t log2Samples = 2,
                 i = 0;
         i < LOG2MAX_ISP-4;
         log2Samples++,
                 i++){
        /// Après la mesure des échantillons, remplacer les
        /// échantillons dans le tableau d'échantillons par
        /// leur logarithme en base 2.
        samples.at(i) = log2Samples;
    }

    // Créer un fichier
    ofstream datafile;
    datafile.open("data/primalityTestsDatasGPU.dat",ios::out);
    if (datafile.bad()){
	cout << "Problème à l'ouverture du fichier" << endl;
    }else {
	for (int i =0; i < samples.size(); i++){
		datafile << samples.at(i) << '\t' << timeMeasurements.at(i) << '\n';
	}
    }
	cout << " Fin Génération. " << endl;

}

void generatePrimeFactorisationDataFileGPU(){
	cout << " Génération des données pour la factorisation sur le GPU " << endl;
    vector<uint64_t> samples = generatePrimeFactorisationSamples();
    vector<float> timeMeasurements = generateGPUPrimeFactorisationMeasurement(samples);

    // Créer un fichier
    ofstream datafile;
    datafile.open("data/factorisationDatasGPU.dat",ios::out);
    if (datafile.bad()){
	cout << "Problème à l'ouverture du fichier" << endl;
    }else {
	for (int i =0; i < samples.size(); i++){
		datafile << samples.at(i) << '\t' << timeMeasurements.at(i) << '\n';
	}
    }
	cout << " Fin Génération. " << endl;
}

/** \brief  Je suis une fonction qui génère des nombres
 *          avec une longeur allant de 4 à 35 bits.
 *  @return vector<uint64_t> Les échantillons.
 */
vector<uint64_t> generatePrimalityTestsSamples() {
    vector<uint64_t> res(0);

    for (   uint64_t currentSample = 0b1000,
            i = 4;
            i < LOG2MAX_ISP;
            currentSample <<= 1,
            i++
    ) {
        res.push_back(currentSample);
    }

    return res;
}

/** \brief  Je suis une fonction qui crée les mesures de tests de temps pour
 *          l'algorithme de tests de primalitée pour un tableau d'échantillons donné.
 *
 *  @param  samples Les échantillons sur lesquels effectuer la mesure.
 *  @return vector<float> Les mesures de temps.
 */
 vector<float> generatePrimalityTestsMeasurement(vector<uint64_t> samples){
     vector<float> res(0);
     for (int i = 0; i < samples.size(); i++){
         ChronoCPU *currentChrono = new ChronoCPU();
         currentChrono->start();
         isPrimeCPU_v0(samples.at(i));
         currentChrono->stop();
         res.push_back(currentChrono->elapsedTime());
         delete currentChrono;
     }
     return res;
 }


/** \brief  Je suis une fonction qui crée les mesures de tests de temps pour
 *          l'algorithme de tests de primalitée pour un tableau d'échantillons donné.
 *
 *  @param  samples Les échantillons sur lesquels effectuer la mesure.
 *  @return vector<float> Les mesures de temps.
 */
 vector<float> generateGPUPrimalityTestsMeasurement(vector<uint64_t> samples){
     vector<float> res(0);
     for (int i = 0; i < samples.size(); i++){
	     //printf("%d round N = %ld\n", i, samples.at(i));
	 /* GPU Routine Alloc */
         uint64_t N = samples.at(i);
	 uint64_t sqrtN = sqrt(N) + 1;
	 uint64_t nombresDePossiblesPremiers = N-2;

	 uint64_t *possibles_premiers = (uint64_t *)malloc(sizeof(uint64_t) * nombresDePossiblesPremiers);
	 for (int i = 0, j = 2; j < N; possibles_premiers[i] =j,i++,j++);
	 unsigned int *res_operations = (unsigned int*)malloc(sizeof(unsigned int) * GRIDDIM(sqrtN));

	 uint64_t *dev_possibles_premiers;
	 hipMalloc((void**)&dev_possibles_premiers, sizeof(uint64_t) * nombresDePossiblesPremiers);
	 unsigned int *dev_res_operations;
	 hipMalloc((void**)&dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN));
	 
	hipMemcpy(dev_possibles_premiers, possibles_premiers, sizeof(uint64_t) * (nombresDePossiblesPremiers), hipMemcpyHostToDevice);
       	hipMemcpy(dev_res_operations, res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyHostToDevice);
	 /* END GPU ROUTINE ALLOC*/
	 ChronoCPU *currentChrono = new ChronoCPU();
        // Start chrono
	 currentChrono->start();
	isPrime<<<GRIDDIM(sqrtN),BLOCKDIM,SIZEMEM>>>(dev_possibles_premiers, dev_res_operations, N, sqrtN); 
	currentChrono->stop(); 
	// end chrono
	
	/* GPU Routine Dealloc */
	 hipMemcpy(res_operations, dev_res_operations, sizeof(unsigned int) * GRIDDIM(sqrtN), hipMemcpyDeviceToHost);
	hipFree(dev_possibles_premiers);
	hipFree(dev_res_operations);
	free(possibles_premiers);
	free(res_operations);
	 /* GPU Routine Dealloc */
	res.push_back(currentChrono->elapsedTime()); //push time 
         delete currentChrono;
     }
     return res;
 }

/** \brief  Je suis une fonction qui aggrège les mesures de
*          nos tests de performances de la recherche de nombre premiers
*          pour une création de graphes ultérieure avec GnuPlot.
* @return boost::tuple<vector<float>,vector<int>> couple de (Tableau de Mesures de temps, Tableau de Logarithme en base 2 des échantillons).
*
boost::tuple<vector<float>,vector<uint64_t>>
createResearchOfPrimesDatas()
{
    vector<uint64_t> limits = generateResearchOfPrimesLimits();
    vector<float> timeMeasurements = generateResearchOfPrimesMeasurement(limits);

    for (uint64_t log2Samples = 2,
                 i = 0;
         i < LOG2MAX_ROP-4;
         log2Samples++,
                 i++){
        /// Après la mesure des échantillons, remplacer les
        /// échantillons dans le tableau d'échantillons par
        /// leur logarithme en base 2.
        limits.at(i) = log2Samples;
    }

    return boost::tuple<vector<float>, vector<uint64_t>>{timeMeasurements,limits};
}
*/

/** \brief  Je suis une fonction qui génère des puissances de 2.
 *  @return vector<uint64_t> Les échantillons.
 */
vector<uint64_t> generateResearchOfPrimesLimits() {
    vector<uint64_t> res(0);

    for (   uint64_t currentSample = 0b1000,
                    i = 4;
            i < LOG2MAX_ROP;
            currentSample <<= 1,
                    i++
            ) {
        res.push_back(currentSample);
    }

    return res;
}

/** \brief  Je suis une fonction qui crée les mesures de tests de temps pour
 *          l'algorithme de tests de primalitée pour un tableau d'échantillons donné.
 *
 *  @param  samples Les échantillons sur lesquels effectuer la mesure.
 *  @return vector<float> Les mesures de temps.
 */
vector<float> generateResearchOfPrimesMeasurement(vector<uint64_t> limits){
    vector<float> res(0);
    for (int i = 0; i < limits.size(); i++){
        ChronoCPU *currentChrono = new ChronoCPU();
        currentChrono->start();
        searchPrimesCPU_v0(limits.at(i));
        currentChrono->stop();
        res.push_back(currentChrono->elapsedTime());
        delete currentChrono;
    }
    return res;
}

/** \brief  Je suis une fonction qui crée les mesures de tests de temps pour
 *          l'algorithme de tests de primalitée pour un tableau d'échantillons donné.
 *
 *  @param  samples Les échantillons sur lesquels effectuer la mesure.
 *  @return vector<float> Les mesures de temps.
 */
vector<float> generateGPUResearchOfPrimesMeasurement(vector<uint64_t> limits){
    vector<float> res(0);
    for (int i = 0; i < limits.size(); i++){
        ChronoCPU *currentChrono = new ChronoCPU();
        currentChrono->start();
        searchPrimesCPU_v0(limits.at(i));
        currentChrono->stop();
        res.push_back(currentChrono->elapsedTime());
        delete currentChrono;
    }
    return res;
}


/** \brief  Je suis une fonction qui aggrège les mesures de
*          nos tests de performances de la recherche de nombre premiers
*          pour une création de graphes ultérieure avec GnuPlot.
* @return boost::tuple<vector<float>,vector<int>> couple de (Tableau de Mesures de temps, Tableau de Logarithme en base 2 des échantillons).
*
boost::tuple<vector<float>,vector<uint64_t>>
createPrimeFactorisationDatas()
{
    vector<uint64_t> samples = generatePrimeFactorisationSamples();
    vector<float> timeMeasurements = generatePrimeFactorisationMeasurement(samples);

    return boost::tuple<vector<float>, vector<uint64_t>>{timeMeasurements,samples};
}
*/

/** \brief  Je suis une fonction qui génère des puissances de 2.
 *  @return vector<uint64_t> Les échantillons.
 */
vector<uint64_t> generatePrimeFactorisationSamples() {
    vector<uint64_t> res(0);
    int limit = 40000;
    double step = limit/25;

    for (
            int i = 10;
            i < limit;
            i+=step){
        res.push_back(i);
    }

    return res;
}

/** \brief  Je suis une fonction qui crée les mesures de tests de temps pour
 *          l'algorithme de tests de primalitée pour un tableau d'échantillons donné.
 *
 *  @param  samples Les échantillons sur lesquels effectuer la mesure.
 *  @return vector<float> Les mesures de temps.
 */
vector<float> generatePrimeFactorisationMeasurement(vector<uint64_t> samples){
    vector<float> res(0);
    for (int i = 0; i < samples.size(); i++){
        ChronoCPU *currentChrono = new ChronoCPU();
        vector<cell> factors(0);
        currentChrono->start();
        factoCPU(samples.at(i), &factors);
        currentChrono->stop();
        res.push_back(currentChrono->elapsedTime());
        delete currentChrono;
    }
    return res;
}

/** \brief  Je suis une fonction qui crée les mesures de tests de temps pour
 *          l'algorithme de tests de primalitée pour un tableau d'échantillons donné.
 *
 *  @param  samples Les échantillons sur lesquels effectuer la mesure.
 *  @return vector<float> Les mesures de temps.
 */
vector<float> generateGPUPrimeFactorisationMeasurement(vector<uint64_t> samples){
    vector<float> res(0);
    for (int i = 0; i < samples.size(); i++){
        ChronoCPU *currentChrono = new ChronoCPU();
        vector<cell> factors(0);
        currentChrono->start();
        factoCPU(samples.at(i), &factors);
        currentChrono->stop();
        res.push_back(currentChrono->elapsedTime());
        delete currentChrono;
    }
    return res;
}
